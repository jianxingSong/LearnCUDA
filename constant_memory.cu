#include "hip/hip_runtime.h"
//
// Created by joey on 2024/8/3.
//
#include "iostream"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"

using namespace std;

__constant__ float c_data;
__constant__ float c_data2 = 6.6f;

__global__ void kernel(){
    printf("Constant data c_data = %.2f.\n", c_data);
}

int main(){
    int deID = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, deID);
    cout << "运行GPU设备：" << deviceProp.name << endl;

    float h_data = 8.8f;
    hipMemcpyToSymbol(HIP_SYMBOL(c_data), &h_data, sizeof(float));

    dim3 grid(1);
    dim3 block(1);
    kernel<<<grid, block>>>();
    hipDeviceSynchronize();
    hipMemcpyFromSymbol(&h_data, HIP_SYMBOL(c_data2), sizeof(float));
    printf("Costant data h_data = %.2f.\n", h_data);

    hipDeviceReset();


    return 0;
}

