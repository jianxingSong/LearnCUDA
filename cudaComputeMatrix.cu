#include "hip/hip_runtime.h"
//
// Created by joey on 2024/6/20.
//

#include "cudaComputeMatrix.cuh"

__global__ void MatrixMulKernel(float* d_A, float* d_B, float* d_C, int M, int N, int K){
    int Row = blockIdx.y * blockDim.y + threadIdx.y;
    int Col = blockIdx.x * blockDim.x + threadIdx.x;

    if((Row < M) && (Col < N)){
        float Cvalue = 0.0;
        for(int k = 0;k < K;++k){
            Cvalue += d_A[Row * K + k] * d_B[k * N * Col];
        }
        d_C[Row * N + Col] = Cvalue;
    }
}

void MatrixMultiply(float* h_A, float* h_B, float* h_C, int M, int N, int K){
    int size_A = M * K * sizeof(float);
    int size_B = K * N * sizeof(float);
    int size_C = M * N * sizeof(float);

    //分配设备内存
    float* d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, size_A);
    hipMalloc((void **)&d_B, size_B);
    hipMalloc((void **)&d_C, size_C);

    //从cpu拷贝矩阵到gpu
    hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);

    //一个核函数分配一个grid，一个grid划分为n个block，每个block里面有很多个线程
    dim3 dimBlock(16, 16); //设置一共有16*16个block
    dim3 dimGrid((N + 16 - 1) / 16, (M + 16 - 1) / 16); //每个block中这么多个线程，要确保每个元素都能对应一个线程

    MatrixMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, M, N, K);

    hipMemcpy(h_C, d_C, size_C, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}