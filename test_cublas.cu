//
// Created by joey on 2024/8/6.
//
#include "iostream"
#include <hipblas.h>
#include "hipblas.h"
#include "thread"

using namespace std;

#define N 1024

int main(){
    float *h_a = (float*)malloc(N*N*sizeof(float));
    float *h_b = (float*)malloc(N*N*sizeof(float));
    float *h_c = (float*)malloc(N*N*sizeof(float));
    float* d_a, *d_b, *d_c;

    // 测速
    cublasInit();
    for(int i = 0;i < N * N;i++){
        h_a[i] = 2;
        h_b[i] = 2;
    }
//    cublasAlloc(N * N, sizeof(float), (void**)&d_a);
//    cublasAlloc(N * N, sizeof(float), (void**)&d_b);
//    cublasAlloc(N * N, sizeof(float), (void**)&d_c);
    hipMalloc((void**)&d_a, sizeof(float) * N * N);
    hipMalloc((void**)&d_b, sizeof(float) * N * N);
    hipMalloc((void**)&d_c, sizeof(float) * N * N);
    size_t size = sizeof(float) * N * N;
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);
    hipMemcpy(d_c, h_c, size, hipMemcpyHostToDevice);
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    float alpha = 1.0;
    float beta = 0.0;
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, d_a, N, d_b, N, &beta, d_c, N);
    hipMemcpy(h_c, d_c, N * N * sizeof(int), hipMemcpyDeviceToHost);

    for(int i = 0;i < N;i++){
        for(int j = 0;j < N;j++){
            cout << h_c[i * N + j] << " ";
        }
        cout << endl;
    }

    free(h_a);
    free(h_b);
    free(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);


    return 0;
}
