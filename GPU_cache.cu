#include "hip/hip_runtime.h"
//
// Created by joey on 2024/8/3.
//
#include "iostream"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"

using namespace std;

__global__ void kernel(void){

}

int main(){
    int deID = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, deID);
    cout << "运行GPU设备：" << deviceProp.name << endl;


    if(deviceProp.globalL1CacheSupported){
        cout << "支持L1缓存" << endl;
    } else {
        cout << "不支持L1缓存" << endl;
    }
    cout << "L2缓存大小：" << deviceProp.l2CacheSize / (1024 * 1024) << "M" << endl;

    return 0;
}

