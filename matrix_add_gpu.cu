#include "hip/hip_runtime.h"
//
// Created by joey on 2024/8/4.
//

#include "iostream"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"

using namespace std;

__global__ void addKernel(int* a, int* b, int* c, const int N){
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if(idx < N * N){
        c[idx] = a[idx] + b[idx];
    }
}

void setValue(int* m, int N){
    for(int i = 0;i < N;i++){
        for(int j = 0;j < N;j++){
            m[i * N + j] = 1;
        }
    }
}

int main(){
    const int N = 512;
    int* h_a = new int[N * N];
    int* h_b = new int[N * N];
    int* h_c = new int[N * N];
    setValue(h_a, N);
    setValue(h_b, N);
    setValue(h_c, N);

    int* d_a, *d_b, *d_c;
    size_t size = N * N * sizeof(int);
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);
    hipMemcpy(d_c, h_c, size, hipMemcpyHostToDevice);

    dim3 block(32);
    dim3 grid((N * N - 1 + 32) / 32);
    addKernel<<<grid, block>>>(d_a, d_b, d_c, N);
    hipDeviceSynchronize();

    hipMemcpy(h_a, d_a, size, hipMemcpyDeviceToHost);
    hipMemcpy(h_b, d_b, size, hipMemcpyDeviceToHost);
    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);
    for(int i = 0;i < N;i++){
        for(int j = 0;j < N;j++){
            cout << h_c[i * N + j] << " ";
        }
        cout << endl;
    }

    delete[] h_a;
    delete[] h_b;
    delete[] h_c;
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);


    return 0;
}