#include "hip/hip_runtime.h"
//
// Created by joey on 2024/8/2.
//
#include "stdlib.h"
#include "iostream"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"

using namespace std;

__device__ int d_x = 1;
__device__ int d_y[2];


__global__ void kernel(void){
    d_y[0] += d_x;
    d_y[1] += d_x;

    printf("d_x = %d, d_y[0] = %d, d_y[1] = %d.\n", d_x, d_y[0], d_y[1]);
}

int main(){
    int h_y[2] = {10, 20};
    hipMemcpyToSymbol(HIP_SYMBOL(d_y), h_y, sizeof(int) * 2);

    dim3 block (1);
    dim3 grid(1);
    kernel<<<grid, block>>>();
    hipDeviceSynchronize();
    hipMemcpyFromSymbol(h_y, HIP_SYMBOL(d_y), sizeof(int) * 2);
    hipDeviceSynchronize();
    printf("h_y[0] = %d, h_y[1] = %d.\n", h_y[0], h_y[1]);


    return 0;
}
