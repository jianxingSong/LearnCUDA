#include "hip/hip_runtime.h"
//
// Created by joey on 2024/8/2.
//
#include "cstdlib"
#include "iostream"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"

using namespace std;

__global__ void kernel(float* d_A, const int N){
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    const int n = bid * blockDim.x + tid;
    __shared__ float s_arr[32];

    if(n < N){
        s_arr[tid] = d_A[n];
    }
    __syncthreads();

    if(tid == 0){
        for(int i = 0;i < 32;++i){
            printf("kernel: %f, blockIdx: %d\n", s_arr[i], bid);
        }
    }
}


int main(){
    int deID = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, deID);
    cout << "运行GPU设备：" << deviceProp.name << endl;

    int nElems = 64;
    int nBytes = nElems * sizeof(float);

    float *hA = nullptr;
    hA = (float*)malloc(nBytes);
    for(int i = 0;i < nElems;i++)
        hA[i] = float(i);
    float *dA = nullptr;
    hipMalloc(&dA, nBytes);
    hipMemcpy(dA, hA, nBytes, hipMemcpyHostToDevice);

    dim3 block(32);
    dim3 grid(2);
    kernel<<<grid, block>>>(dA, 64);
    hipFree(dA);
    free(hA);


    return 0;
}